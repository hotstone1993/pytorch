#include "hip/hip_runtime.h"
#include "my_shift_kernel.cuh"
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/KernelUtils.h>

using namespace at::cuda::detail;

__global__ void gup_shift(float* input, float* output, int64_t move) {
    // ...
}

void cpu_shift(float* input, float* ouput, int64_t move) {
    // ...
}


torch::Tensor shift_op(torch::Tensor input, int64_t move) {
    torch::Device device(torch::kCUDA, 0);
    torch::Tensor output = torch::zeros(input.size(0), torch::kFloat);
    output = output.to(input);

    if (input.device() == device){
        const hipStream_t stream = at::cuda::getCurrentCUDAStream();

        const dim3 grid(GET_BLOCKS(input.size(0)));
        const dim3 block(CUDA_NUM_THREADS);
        
        gup_shift<<<grid, block, 0, stream>>>(input.data_ptr<float>(), output.data_ptr<float>(), move);
    } else {
        cpu_shift(input.data_ptr<float>(), output.data_ptr<float>(), move);
    }

    return output;
}